#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <iostream>

using namespace std;
using namespace cv;

#define COLS 3
#define ROWS 3

extern "C"    void iteration_mat1()
{
    hipfftComplex *result_temp_din = (hipfftComplex*)malloc(COLS*ROWS * sizeof(hipfftComplex));
    hipfftHandle p;
    //输入赋值数据
    for (size_t j = 0; j < ROWS; j++)
    {
        for (size_t i = 0; i < COLS; i++)
        {
            result_temp_din[i + j*COLS].x = (i + 1)*(j + 1);
            cout << result_temp_din[i + j*COLS].x << " ";
            result_temp_din[i + j*COLS].y = 0;
        }
    }
    cout << endl;

    size_t pitch;

    hipfftComplex *t_result_temp_din;
    hipMallocPitch((void**)&t_result_temp_din, &pitch, COLS * sizeof(hipfftComplex), ROWS);
     
    hipfftComplex *t_result_temp_out;
    hipMallocPitch((void**)&t_result_temp_out, &pitch, COLS * sizeof(hipfftComplex), ROWS);

    //将值辅到Device
    //hipMemcpy2D(t_result_temp_din, pitch, result_temp_din, COLS * sizeof(hipfftComplex), COLS * sizeof(hipfftComplex), ROWS, hipMemcpyHostToDevice);
    hipMemcpy(t_result_temp_din,result_temp_din,  ROWS * sizeof(hipfftComplex)* COLS, hipMemcpyHostToDevice);

    //forward fft  制定变换规则
    hipfftPlan2d(&p, ROWS, COLS, HIPFFT_C2C);

    //执行变换
    hipfftExecC2C(p, (hipfftComplex*)t_result_temp_din, (hipfftComplex*)t_result_temp_out, HIPFFT_FORWARD);

    //将值辅到host
    hipMemcpy(result_temp_din,  t_result_temp_out, ROWS * sizeof(hipfftComplex)* COLS, hipMemcpyDeviceToHost);
    //hipMemcpy2D(result_temp_din, pitch, t_result_temp_out, COLS * sizeof(hipfftComplex), sizeof(hipfftComplex)* ROWS, COLS, hipMemcpyDeviceToHost);


    //提取实部和虚部
    for (size_t j = 0; j < ROWS; j++)
    {
        for (size_t i = 0; i < COLS; i++)
        {
            cout << result_temp_din[i + j*COLS].x << " ";//实部  
            cout << result_temp_din[i + j*COLS].y << endl;//虚部  
        }
    }

}